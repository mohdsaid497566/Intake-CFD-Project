
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello from CUDA!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
